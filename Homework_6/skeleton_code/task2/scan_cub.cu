#include "scan_cub.h"
#include "../include/utils.h"
#include <cub/device/device_scan.cuh>

ScanCub::ScanCub() { }

ScanCub::~ScanCub() {
    CUDA_CHECK(hipFree(bufferDev_));
}

void ScanCub::inclusiveSum(const int *inDev, int *outDev, int N) {
    size_t size = 0;
    CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(nullptr, size, inDev, outDev, N));

    if (bufferSize_ < size) {
        CUDA_CHECK(hipFree(bufferDev_));
        CUDA_CHECK(hipMalloc(&bufferDev_, size));
        bufferSize_ = size;
    }

    CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(bufferDev_, size, inDev, outDev, N));
}
