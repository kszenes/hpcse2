#include "scan.h"
#include "../include/utils.h"
#include <cub/device/device_scan.cuh>

Scan::Scan() { }

Scan::~Scan() {
    CUDA_CHECK(hipFree(bufferDev_));
}

void Scan::inclusiveSum(const int *inDev, int *outDev, int N) {
    size_t size = 0;
    CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(nullptr, size, inDev, outDev, N));

    if (bufferSize_ < size) {
        CUDA_CHECK(hipFree(bufferDev_));
        CUDA_CHECK(hipMalloc(&bufferDev_, size));
        bufferSize_ = size;
    }

    CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(bufferDev_, size, inDev, outDev, N));
}
