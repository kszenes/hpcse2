#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <cassert>
#include <limits>

#define FULL_MASK 0xffffffff

struct Pair {
    double max;
    int idx;
};

/// Find the maximum value `a` among all warps and return {max value, index of
/// the max}. The result must be correct on at least the 0th thread of each warp.
__device__ Pair argMaxWarp(double a) {
    // TODO: 1.b) Compute the argmax of the given value.
    //            Return the maximum and the location of the maximum (0..31).
    Pair result;
    result.max = a;
    result.idx = threadIdx.x & 31;
    int idx = result.idx;
    double max;
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        max = __shfl_down_sync(FULL_MASK, result.max, offset);
        idx = __shfl_down_sync(FULL_MASK, result.idx, offset);
        if (max > result.max) {
            result.idx = idx;
            result.max = max;
        }

    }
    return result;
}


/// Returns the argmax of all values `a` within a block,
/// with the correct answer returned at least by the 0th thread of a block.
__device__ Pair argMaxBlock(double a) {
    // TODO: 1.c) Compute the argmax of the given value.
    //            Return the maximum and the location of the maximum (0..1023).
    // NOTE: For 1.c) implement either this or `sumBlock`!
    Pair result;
    result.max = 0.0;
    result.idx = 0;

    // ...

    return result;
}


void argMax1M(const double * /* aDev */, Pair * /* bDev */, int N) {
    assert(N <= 1024 * 1024);
    // TODO: 1.d) Implement either this or `sum1M`.
    //            Avoid copying any data back to the host.
    //            Hint: The solution requires more CUDA operations than just
    //            calling a single kernel. Feel free to use whatever you find
    //            necessary.
}

#include "_reduction_argmax.h"

int main() {
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 3);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 32);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 320);
    testSmallArgMax(argMaxWarpTestKernel, argMaxWarpCheck, 32, 1023123);
    printf("argMaxWarp OK.\n");

    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 32);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 1024);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 12341);
    testSmallArgMax(argMaxBlockTestKernel, argMaxBlockCheck, 1024, 1012311);
    printf("argMaxBlock OK.\n");

    testLargeArgMax("argMax1M", argMax1M, 32);
    testLargeArgMax("argMax1M", argMax1M, 1024);
    testLargeArgMax("argMax1M", argMax1M, 12341);
    testLargeArgMax("argMax1M", argMax1M, 1012311);
    printf("argMax1M OK.\n");
}
