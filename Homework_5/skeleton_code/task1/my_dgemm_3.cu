#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "utils.h"

#define BLOCK_SIZE 16
#define SHMEM_SIZE (16 * 16)

__global__ void sharedDgemm(
    const int m,
    const int n,
    const int k,
    const double alpha,
    const double* const A,
    const double* const B,
    const double beta,
    double* const C)
{
//   TODO
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int row = blockIdx.x * blockDim.x + tx;
    const int col = blockIdx.y * blockDim.y + ty;

    __shared__ double a[SHMEM_SIZE];
    __shared__ double b[SHMEM_SIZE];

    double tmp = 0;
    
#pragma unroll
    for (int i = 0; i < (k / BLOCK_SIZE); i++){

      a[tx + ty * BLOCK_SIZE] = A[row + (i*BLOCK_SIZE + ty) * m];

      b[tx + ty * BLOCK_SIZE] = B[col*k + i*BLOCK_SIZE + tx];
      __syncthreads();

#pragma unroll
      for (int j = 0; j < BLOCK_SIZE; j++) {
        tmp += a[j*BLOCK_SIZE + tx] * b[ty*BLOCK_SIZE + j];
      }
      __syncthreads();

    }
    C[col*m + row] = alpha*tmp + beta*C[col*m + row];

}

void myDgemm(
    const int m,
    const int n,
    const int k,
    const double alpha,
    const double* const A,
    const double* const B,
    const double beta,
    double* const C)
{
//  TODO
  dim3 dimGrid(k/BLOCK_SIZE, m/BLOCK_SIZE);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  sharedDgemm<<<dimGrid, dimBlock>>>(m, n, k, alpha, A, B, beta, C);
  CUDA_CHECK(hipGetLastError());
}
