#include <hip/hip_runtime.h>

__global__ void computeForcesKernel(int N, const double3 *p, double3 *f) {
  
    for(int idx = blockIdx.x * blockDim.x + threadIdx.x;
            idx < N;
            idx += gridDim.x * blockDim.x){


        f[idx] = double3{0.0, 0.0, 0.0};

        double px = p[idx].x; 
        double py = p[idx].y;
        double pz = p[idx].z;

        double fx = f[idx].x; 
        double fy = f[idx].y;
        double fz = f[idx].z;
        for (int i = 0; i < idx; ++i) {
                double dx = p[i].x - px;
                double dy = p[i].y - py;
                double dz = p[i].z - pz;
                double r = sqrt(dx * dx + dy * dy + dz * dz);
                double inv_r = 1.0 / r;
                fx += dx * inv_r * inv_r * inv_r;
                fy += dy * inv_r * inv_r * inv_r;
                fz += dz * inv_r * inv_r * inv_r;
        }

        for (int i = idx+1; i < N; ++i) {
                double dx = p[i].x - px;
                double dy = p[i].y - py;
                double dz = p[i].z - pz;
                double r = sqrt(dx * dx + dy * dy + dz * dz);
                double inv_r = 1.0 / r;
                fx += dx * inv_r * inv_r * inv_r;
                fy += dy * inv_r * inv_r * inv_r;
                fz += dz * inv_r * inv_r * inv_r;
        }
        f[idx] = double3{fx, fy, fz};
    }
}

void computeForces(int N, const double3 *p, double3 *f) {
    constexpr int numThreads = 1024;
    int numBlocks = (N + numThreads - 1) / numThreads;
    computeForcesKernel<<<numBlocks, numThreads>>>(N, p, f);
}
